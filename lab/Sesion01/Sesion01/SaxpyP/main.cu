
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void saxpyP (int N, float a, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  y[i] = a * x[i] + y[i];
}

void InitV(int N, float *v);
int TestSaxpy(int N, float a, float *x, float *y, float *Y);
void CheckCudaError(char sms[], int line);



int main(int argc, char** argv)

{
  unsigned int N;
  unsigned int numBytes;
  unsigned int nBlocks, nThreads;
 
  hipEvent_t E0, E1, E2, E3, E4, E5;
  float TiempoTotal, TiempoKernel;

  float *h_x, *h_y, *H_y;
  float *d_x, *d_y;

  if (argc==2)
  {
     N = atoi(argv[1]);
  }
  else if (argc==3)
  {
     N = atoi(argv[1]);
     nThreads = atoi(argv[2]);
  }
  else if (argc>3) 
  {
     printf("Command line: %s [N [nThreads]]\n", argv[0]);
     exit(EXIT_FAILURE);
  } 
  else {
     N = 1024 * 1024 * 16;
     nThreads = 1024;
     printf("Default N, nThreads\n");
  } 

  printf("N = %d nThreads=%d\n", N, nThreads);
  nBlocks = N/nThreads;  // Solo funciona bien si N multiplo de nThreads
  numBytes = N * sizeof(float);


  hipEventCreate(&E0);
  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);
  hipEventCreate(&E4);
  hipEventCreate(&E5);

  // Obtener Memoria en el host
  h_x = (float*) malloc(numBytes); 
  h_y = (float*) malloc(numBytes); 
  H_y = (float*) malloc(numBytes);    // Solo se usa para comprobar el resultado

  // Obtiene Memoria [pinned] en el host
  //cudaMallocHost((float**)&h_x, numBytes); 
  //cudaMallocHost((float**)&h_y, numBytes); 
  //cudaMallocHost((float**)&H_y, numBytes);   // Solo se usa para comprobar el resultado

  // Inicializa los vectores
  InitV(N, h_x);
  InitV(N, h_y);

 
  hipEventRecord(E0, 0);
  hipEventSynchronize(E0);
 
  // Obtener Memoria en el device
  hipMalloc((float**)&d_x, numBytes); 
  hipMalloc((float**)&d_y, numBytes); 
  CheckCudaError((char *) "Obtener Memoria en el device", __LINE__); 

  hipEventRecord(E1, 0);
  hipEventSynchronize(E1);

  // Copiar datos desde el host en el device 
  hipMemcpy(d_x, h_x, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, numBytes, hipMemcpyHostToDevice);
  CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);

  hipEventRecord(E2, 0);
  hipEventSynchronize(E2);

  // Ejecutar el kernel 
  saxpyP<<<nBlocks, nThreads>>>(N, 3.5, d_x, d_y);
  CheckCudaError((char *) "Invocar Kernel", __LINE__);

  hipEventRecord(E3, 0);
  hipEventSynchronize(E3);

  // Obtener el resultado desde el host 
  // Guardamos el resultado en H_y para poder comprobar el resultado
  hipMemcpy(H_y, d_y, numBytes, hipMemcpyDeviceToHost); 
  CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

  hipEventRecord(E4, 0);
  hipEventSynchronize(E4);

  // Liberar Memoria del device 
  hipFree(d_x); hipFree(d_y);

  hipDeviceSynchronize();

  hipEventRecord(E5, 0);
  hipEventSynchronize(E5);

  hipEventElapsedTime(&TiempoTotal,  E0, E5);
  hipEventElapsedTime(&TiempoKernel, E2, E3);
 
  printf("nThreads: %d\n", nThreads);
  printf("nBlocks: %d\n", nBlocks);

  printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
  printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);

  hipEventDestroy(E0); 
  hipEventDestroy(E1); 
  hipEventDestroy(E2); 
  hipEventDestroy(E3); 
  hipEventDestroy(E4); 
  hipEventDestroy(E5);

  if (TestSaxpy(N, 3.5, h_x, h_y, H_y))
    printf ("TEST PASS\n");
  else
    printf ("TEST FAIL\n");

}


void InitV(int N, float *v) {
   int i;
   for (i=0; i<N; i++) 
     v[i] = rand();
   
}
int error(float a, float b) {

  if (abs (a - b) / a > 0.000001) return 1;
  else  return 0;

}

int TestSaxpy(int N, float a, float *x, float *y, float *Y) {
   int i;
   float tmp;

   for (i=0; i<N; i++) {
     tmp = a * x[i] + y[i];
     if (error(tmp, Y[i])) {
       printf ("%d: %f - %f = %f \n", i, tmp, Y[i], tmp - Y[i]);
       return 0;
     }
   }
   return 1;
}

void CheckCudaError(char sms[], int line) {
  hipError_t error;
 
  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }


}


